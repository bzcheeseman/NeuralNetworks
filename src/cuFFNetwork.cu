#include "hip/hip_runtime.h"
//
// Created by Aman LaChapelle on 9/21/16.
//
// NeuralNetworks
// Copyright (C) 2016  Aman LaChapelle
//
// Full license at NeuralNetworks/LICENSE.txt
//

/*
    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "../include/cuFFNetwork.hpp"

/*******************************************
 * cuLayer
 *******************************************/

cuLayer::cuLayer(int in, int out) : in(in), out(out) {
  w = Eigen::MatrixXf(out, in);
  b = Eigen::VectorXf(out);
  z = Eigen::VectorXf(out);
  a = Eigen::VectorXf(out);

  checkCudaErrors(hipSetDevice(0));

  float *devicedata;
  float mean = (float)0.0;
  float stddev = (float)(1.0/(float)sqrt( (float)in ));

  hiprandGenerator_t gen;
  //create generator
  checkCurandErrors(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
  //set generator seed
  auto now = std::chrono::high_resolution_clock::now();
  std::uint64_t nanos = std::chrono::duration_cast<std::chrono::nanoseconds>(now.time_since_epoch()).count();
  checkCurandErrors(hiprandSetPseudoRandomGeneratorSeed(gen, nanos)); //set seed here

  checkCudaErrors(hipMalloc(&devicedata, (in*out)*sizeof(float))); //malloc size of weights
  checkCurandErrors(hiprandGenerateNormal(gen, devicedata, (std::size_t)(in*out + (in*out)%2), mean, stddev)); //generate numbers
  checkCudaErrors(hipMemcpy(w.data(), devicedata, (in*out)*sizeof(float), hipMemcpyDeviceToHost)); //copy it back
  checkCudaErrors(hipFree(devicedata)); //free pointer to realloc

  checkCudaErrors(hipMalloc((void **)&devicedata, (out)*sizeof(float))); //realloc for biases
  checkCurandErrors(hiprandGenerateNormal(gen, devicedata, (std::size_t)(out + out%2), mean, stddev)); //generate numbers
  checkCudaErrors(hipMemcpy(b.data(), devicedata, (out)*sizeof(float), hipMemcpyDeviceToHost)); //copy it back

  float *dev_z, *dev_a;

  checkCudaErrors(hipMalloc(&dev_z, out * sizeof(float)));
  checkCudaErrors(hipMemset(dev_z, 0.0f, out*sizeof(float)));
  checkCudaErrors(hipMemcpy(z.data(), dev_z, out*sizeof(float), hipMemcpyDeviceToHost));

  checkCudaErrors(hipMalloc(&dev_a, out * sizeof(float)));
  checkCudaErrors(hipMemset(dev_a, 0.0f, out*sizeof(float)));
  checkCudaErrors(hipMemcpy(a.data(), dev_a, out*sizeof(float), hipMemcpyDeviceToHost));


  checkCudaErrors(hipFree(devicedata)); //free pointer
  checkCudaErrors(hipFree(dev_z));
  checkCudaErrors(hipFree(dev_a));
  checkCurandErrors(hiprandDestroyGenerator(gen));

  checkCudaErrors(hipDeviceSynchronize());
}

std::ostream &operator<<(std::ostream &out, cuLayer &layer) {
  out << "Inputs: " << layer.in << " Outputs: " << layer.out << std::endl;
  out << "==========Weights==========\n" << layer.w << std::endl;
  out << "\n==========Bias==========\n"<< layer.b << std::endl;
  out << "\n==========Z==========\n"<< layer.z << std::endl;
  out << "\n==========Activations==========\n"<< layer.a << std::endl;
  return out;
}

void cuLayer::copy_to_device() {

  checkCudaErrors(hipMalloc(&dev_w, in*out * sizeof(float)));
  checkCudaErrors(hipMemcpyAsync(dev_w, &w.data()[0], in*out * sizeof(float), hipMemcpyHostToDevice));

  checkCudaErrors(hipMalloc(&dev_b, out * sizeof(float)));
  checkCudaErrors(hipMemcpyAsync(dev_b, &b.data()[0], out * sizeof(float), hipMemcpyHostToDevice));

  checkCudaErrors(hipMalloc(&dev_z, out * sizeof(float)));
  checkCudaErrors(hipMemcpyAsync(dev_z, &z.data()[0], out * sizeof(float), hipMemcpyHostToDevice));

  checkCudaErrors(hipMalloc(&dev_a, out * sizeof(float)));
  checkCudaErrors(hipMemcpyAsync(dev_a, &a.data()[0], out * sizeof(float), hipMemcpyHostToDevice));

}

void cuLayer::copy_from_device() {

  checkCudaErrors(hipMemcpyAsync(w.data(), dev_w, out*in*sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpyAsync(b.data(), dev_b, out*sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpyAsync(z.data(), dev_z, out*sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpyAsync(a.data(), dev_a, out*sizeof(float), hipMemcpyDeviceToHost));

}

void cuLayer::free_device_ptr() {

  checkCudaErrors(hipFree(dev_w));
  checkCudaErrors(hipFree(dev_b));
  checkCudaErrors(hipFree(dev_z));
  checkCudaErrors(hipFree(dev_a));

}

void cuLayer::feedThroughLayer(float *device_ptr_input, int len, int batchSize, hipblasHandle_t cublasHandle, hipdnnHandle_t cudnnHandle) {
  assert(len == in);

  float one = 1.0f, zero = 0.0f;

  float *ones;
  checkCudaErrors(hipMalloc(&ones, batchSize * sizeof(float)));
  checkCudaErrors(hipMemset(ones, 1.0f, batchSize * sizeof(float)));

  checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                              out, batchSize, out,
                              &one, dev_w, out, device_ptr_input, in,
                              &zero, dev_z, out));

  checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                              out, batchSize, 1,
                              &one, dev_b, out, ones, 1,
                              &one, dev_z, out));

  checkCUDNN(hipdnnActivationForward(cudnnHandle, activation,
                                    &one, zs, dev_z,
                                    &zero, as, dev_a));
}


/*******************************************
 * cuFFNetwork
 *******************************************/

cuFFNetwork::cuFFNetwork(int gpuid, int batchSize, cuLayer& hidden_layer, cuLayer& output_layer):
        gpuid(gpuid), batchSize(batchSize), hidden_layer(hidden_layer), output_layer(output_layer) {

  //set up device
  checkCudaErrors(hipSetDevice(gpuid));
  checkCudaErrors(hipblasCreate(&cublasHandle));
  checkCUDNN(hipdnnCreate(&cudnnHandle));

  activation_func = Tanh;

  checkCUDNN(hipdnnCreateTensorDescriptor(&input_data)); // init tensor for input data
  checkCUDNN(hipdnnCreateTensorDescriptor(&(hidden_layer.weight))); // init weight tensor for hidden layer
  checkCUDNN(hipdnnCreateTensorDescriptor(&(hidden_layer.bias))); // init bias tensor for hidden layer
  checkCUDNN(hipdnnCreateTensorDescriptor(&(hidden_layer.zs))); // init z tensor for hiddens
  checkCUDNN(hipdnnCreateTensorDescriptor(&(hidden_layer.as))); // init a tensor for hiddens

  checkCUDNN(hipdnnCreateTensorDescriptor(&(output_layer.weight))); // init weight tensor for output layer
  checkCUDNN(hipdnnCreateTensorDescriptor(&(output_layer.bias))); // init bias tensor for output layer
  checkCUDNN(hipdnnCreateTensorDescriptor(&(output_layer.zs))); // init z tensor for hiddens
  checkCUDNN(hipdnnCreateTensorDescriptor(&(output_layer.as))); // init a tensor for hiddens

  checkCUDNN(hipdnnCreateActivationDescriptor(&(hidden_layer.activation))); // init hidden layer activations
  checkCUDNN(hipdnnCreateActivationDescriptor(&(output_layer.activation))); // init output layer activations

  checkCUDNN(hipdnnSetTensor4dDescriptor(hidden_layer.weight, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchSize, hidden_layer.out, 1, 1));
  checkCUDNN(hipdnnSetTensor4dDescriptor(hidden_layer.bias, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchSize, hidden_layer.out, 1, 1));
  checkCUDNN(hipdnnSetTensor4dDescriptor(hidden_layer.zs, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchSize, hidden_layer.out, 1, 1));
  checkCUDNN(hipdnnSetTensor4dDescriptor(hidden_layer.as, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchSize, hidden_layer.out, 1, 1));

  checkCUDNN(hipdnnSetTensor4dDescriptor(output_layer.weight, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchSize, output_layer.out, 1, 1));
  checkCUDNN(hipdnnSetTensor4dDescriptor(output_layer.bias, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchSize, output_layer.out, 1, 1));
  checkCUDNN(hipdnnSetTensor4dDescriptor(output_layer.zs, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchSize, output_layer.out, 1, 1));
  checkCUDNN(hipdnnSetTensor4dDescriptor(output_layer.as, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchSize, output_layer.out, 1, 1));

  if (activation_func == ReLU){
    checkCUDNN(hipdnnSetActivationDescriptor(hidden_layer.activation, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0.0));
  }
  else if (activation_func == Tanh){
    checkCUDNN(hipdnnSetActivationDescriptor(hidden_layer.activation, HIPDNN_ACTIVATION_TANH, HIPDNN_PROPAGATE_NAN, 0.0));
  }
  else if (activation_func == Sigmoid){
    checkCUDNN(hipdnnSetActivationDescriptor(hidden_layer.activation, HIPDNN_ACTIVATION_SIGMOID, HIPDNN_PROPAGATE_NAN, 0.0));
  }


  checkCUDNN(hipdnnSetActivationDescriptor(output_layer.activation, HIPDNN_ACTIVATION_SIGMOID, HIPDNN_PROPAGATE_NAN, 0.0));

  checkCUDNN(hipdnnCreateOpTensorDescriptor(&mult));
  checkCUDNN(hipdnnCreateOpTensorDescriptor(&add));

  checkCUDNN(hipdnnSetOpTensorDescriptor(mult, HIPDNN_OP_TENSOR_MUL, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN));
  checkCUDNN(hipdnnSetOpTensorDescriptor(add, HIPDNN_OP_TENSOR_ADD, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN));

}

cuFFNetwork::~cuFFNetwork() {

  checkCudaErrors(hipSetDevice(gpuid));
  checkCudaErrors(hipblasDestroy(cublasHandle));
  checkCUDNN(hipdnnDestroy(cudnnHandle));

  checkCUDNN(hipdnnDestroyTensorDescriptor(input_data));

}

Eigen::VectorXf cuFFNetwork::feedForward(float *data) {
  checkCudaErrors(hipSetDevice(gpuid));

  hidden_layer.copy_to_device();

  output_layer.copy_to_device();

  //Copy data to device to pass to the hidden layer
  float *dev_data;
  checkCudaErrors(hipMalloc(&dev_data, hidden_layer.in * sizeof(float)));
  checkCudaErrors(hipMemcpy(dev_data, &data[0], batchSize*hidden_layer.in * sizeof(float), hipMemcpyHostToDevice));

  //hidden layer - feed through
  hidden_layer.feedThroughLayer(dev_data, hidden_layer.in, batchSize, cublasHandle, cudnnHandle);

  //output layer - feed through
  output_layer.feedThroughLayer(hidden_layer.dev_a, output_layer.in, batchSize, cublasHandle, cudnnHandle);

  //copy hidden data back to host
  hidden_layer.copy_from_device();
  //free device pointers
  hidden_layer.free_device_ptr();

  //copy output data back to host
  output_layer.copy_from_device();
  //free device pointers
  output_layer.free_device_ptr();

  checkCudaErrors(hipFree(dev_data));

  checkCudaErrors(hipDeviceSynchronize());

  return output_layer.a;
}