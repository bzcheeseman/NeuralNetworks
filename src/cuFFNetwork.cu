#include "hip/hip_runtime.h"
//
// Created by Aman LaChapelle on 9/21/16.
//
// NeuralNetworks
// Copyright (C) 2016  Aman LaChapelle
//
// Full license at NeuralNetworks/LICENSE.txt
//

/*
    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "../include/cuFFNetwork.hpp"

static inline unsigned int RoundUp(unsigned int numerator, unsigned int denominator)
{
  return (numerator + denominator - 1) / denominator;
}


/*******************************************
 * cuLayer
 *******************************************/

cuFFLayer::cuFFLayer(int in, int out, int gpuid) : in(in), out(out), gpuid(gpuid) {
  w = Eigen::MatrixXf(out, in);
  b = Eigen::VectorXf(out);
  z = Eigen::VectorXf(out);
  a = Eigen::VectorXf(out);

  checkCudaErrors(hipSetDevice(gpuid));

  float *devicedata;
  float mean = (float)0.0;
  float stddev = (float)(1.0/(float)sqrt( (float)in ));

  hiprandGenerator_t gen;
  //create generator
  checkCurandErrors(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
  //set generator seed
  auto now = std::chrono::high_resolution_clock::now();
  std::uint64_t nanos = std::chrono::duration_cast<std::chrono::nanoseconds>(now.time_since_epoch()).count();
  checkCurandErrors(hiprandSetPseudoRandomGeneratorSeed(gen, nanos)); //set seed here

  checkCudaErrors(hipMalloc(&devicedata, (in*out)*sizeof(float))); //malloc size of weights
  checkCurandErrors(hiprandGenerateNormal(gen, devicedata, (std::size_t)(in*out + (in*out)%2), mean, stddev)); //generate numbers
  checkCudaErrors(hipMemcpy(w.data(), devicedata, (in*out)*sizeof(float), hipMemcpyDeviceToHost)); //copy it back
  checkCudaErrors(hipFree(devicedata)); //free pointer to realloc

  checkCudaErrors(hipMalloc((void **)&devicedata, (out)*sizeof(float))); //realloc for biases
  checkCurandErrors(hiprandGenerateNormal(gen, devicedata, (std::size_t)(out + out%2), mean, stddev)); //generate numbers
  checkCudaErrors(hipMemcpy(b.data(), devicedata, (out)*sizeof(float), hipMemcpyDeviceToHost)); //copy it back

  float *dev_z, *dev_a;

  checkCudaErrors(hipMalloc(&dev_z, out * sizeof(float)));
  checkCudaErrors(hipMemset(dev_z, 0.0f, out*sizeof(float)));
  checkCudaErrors(hipMemcpy(z.data(), dev_z, out*sizeof(float), hipMemcpyDeviceToHost));

  checkCudaErrors(hipMalloc(&dev_a, out * sizeof(float)));
  checkCudaErrors(hipMemset(dev_a, 0.0f, out*sizeof(float)));
  checkCudaErrors(hipMemcpy(a.data(), dev_a, out*sizeof(float), hipMemcpyDeviceToHost));


  checkCudaErrors(hipFree(devicedata)); //free pointer
  checkCudaErrors(hipFree(dev_z));
  checkCudaErrors(hipFree(dev_a));
  checkCurandErrors(hiprandDestroyGenerator(gen));

  checkCudaErrors(hipDeviceSynchronize());
}

cuFFLayer::~cuFFLayer() {
  checkCudaErrors(hipSetDevice(gpuid));
  checkCUDNN(hipdnnDestroyTensorDescriptor(layerTensor));
  checkCUDNN(hipdnnDestroyActivationDescriptor(activation));
}

void cuFFLayer::initTensor(int batchSize) {

  checkCudaErrors(hipSetDevice(gpuid));

  checkCUDNN(hipdnnCreateTensorDescriptor(&(layerTensor))); // init tensor for this layer

  checkCUDNN(hipdnnSetTensor4dDescriptor(layerTensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchSize, out, 1, 1));

}

void cuFFLayer::setActivation(hipdnnActivationMode_t cudnnActivationFunc) {
  checkCudaErrors(hipSetDevice(gpuid));

  checkCUDNN(hipdnnCreateActivationDescriptor(&(activation)));
  checkCUDNN(hipdnnSetActivationDescriptor(activation, cudnnActivationFunc, HIPDNN_PROPAGATE_NAN, 0.0));
}

void cuFFLayer::copy_to_device() {

  checkCudaErrors(hipSetDevice(gpuid));

  checkCudaErrors(hipMalloc(&dev_w, in*out * sizeof(float)));
  checkCudaErrors(hipMemcpyAsync(dev_w, &w.data()[0], in*out * sizeof(float), hipMemcpyHostToDevice));

  checkCudaErrors(hipMalloc(&dev_b, out * sizeof(float)));
  checkCudaErrors(hipMemcpyAsync(dev_b, &b.data()[0], out * sizeof(float), hipMemcpyHostToDevice));

  checkCudaErrors(hipMalloc(&dev_z, out * sizeof(float)));
  checkCudaErrors(hipMemcpyAsync(dev_z, &z.data()[0], out * sizeof(float), hipMemcpyHostToDevice));

  checkCudaErrors(hipMalloc(&dev_a, out * sizeof(float)));
  checkCudaErrors(hipMemcpyAsync(dev_a, &a.data()[0], out * sizeof(float), hipMemcpyHostToDevice));

}

void cuFFLayer::copy_from_device() {

  checkCudaErrors(hipSetDevice(gpuid));

  checkCudaErrors(hipMemcpyAsync(w.data(), dev_w, out*in*sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpyAsync(b.data(), dev_b, out*sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpyAsync(z.data(), dev_z, out*sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpyAsync(a.data(), dev_a, out*sizeof(float), hipMemcpyDeviceToHost));

}

void cuFFLayer::free_device_ptr() {

  checkCudaErrors(hipSetDevice(gpuid));

  checkCudaErrors(hipFree(dev_w));
  checkCudaErrors(hipFree(dev_b));
  checkCudaErrors(hipFree(dev_z));
  checkCudaErrors(hipFree(dev_a));

}

void cuFFLayer::feedThroughLayer(float *device_ptr_input, int len, int batchSize, hipblasHandle_t cublasHandle, hipdnnHandle_t cudnnHandle) {
  assert(len == in);

  checkCudaErrors(hipSetDevice(gpuid));

  float one = 1.0f, zero = 0.0f;

  float *ones;
  checkCudaErrors(hipMalloc(&ones, batchSize * sizeof(float)));
  checkCudaErrors(hipMemset(ones, 1.0f, batchSize * sizeof(float)));

  checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                              out, batchSize, out,
                              &one, dev_w, out, device_ptr_input, in,
                              &zero, dev_z, out));

  checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                              out, batchSize, 1,
                              &one, dev_b, out, ones, 1,
                              &one, dev_z, out));

  checkCUDNN(hipdnnActivationForward(cudnnHandle, activation,
                                    &one, layerTensor, dev_z,
                                    &zero, layerTensor, dev_a)); //apply activation within the layer, before giving away output
}

std::ostream &operator<<(std::ostream &out, cuFFLayer &layer) {
  out << "Inputs: " << layer.in << " Outputs: " << layer.out << std::endl;
  out << "==========Weights==========\n" << layer.w << std::endl;
  out << "\n==========Bias==========\n"<< layer.b << std::endl;
  out << "\n==========Z==========\n"<< layer.z << std::endl;
  out << "\n==========Activations==========\n"<< layer.a << std::endl;
  return out;
}


/*******************************************
 * cuFFNetwork
 *******************************************/

cuFFNetwork::cuFFNetwork(int gpuid, int batchSize, cuFFLayer& hidden_layer, cuFFLayer& output_layer):
        gpuid(gpuid), batchSize(batchSize), hidden_layer(hidden_layer), output_layer(output_layer) {

  //set up device
  checkCudaErrors(hipSetDevice(gpuid));

  checkCudaErrors(hipblasCreate(&cublasHandle));
  checkCUDNN(hipdnnCreate(&cudnnHandle));

  activation_func = Tanh;

  checkCUDNN(hipdnnCreateTensorDescriptor(&input_data)); // init tensor for input data

  this->hidden_layer.initTensor(batchSize);

  this->output_layer.initTensor(batchSize);

  if (activation_func == ReLU){
    this->hidden_layer.setActivation(HIPDNN_ACTIVATION_RELU);
  }
  else if (activation_func == Tanh){
    this->hidden_layer.setActivation(HIPDNN_ACTIVATION_TANH);
  }
  else if (activation_func == Sigmoid){
    this->hidden_layer.setActivation(HIPDNN_ACTIVATION_SIGMOID);
  }


  this->output_layer.setActivation(HIPDNN_ACTIVATION_SIGMOID);

}

cuFFNetwork::~cuFFNetwork() {

  checkCudaErrors(hipSetDevice(gpuid));

  checkCudaErrors(hipblasDestroy(cublasHandle));
  checkCUDNN(hipdnnDestroy(cudnnHandle));

  checkCUDNN(hipdnnDestroyTensorDescriptor(input_data));

}

Eigen::VectorXf cuFFNetwork::feedForward(float *data) {
  checkCudaErrors(hipSetDevice(gpuid));

  hidden_layer.copy_to_device();

  output_layer.copy_to_device();

  //Copy data to device to pass to the hidden layer
  float *dev_data;
  checkCudaErrors(hipMalloc(&dev_data, hidden_layer.in * sizeof(float)));
  checkCudaErrors(hipMemcpy(dev_data, &data[0], batchSize*hidden_layer.in * sizeof(float), hipMemcpyHostToDevice));

  //hidden layer - feed through
  hidden_layer.feedThroughLayer(dev_data, hidden_layer.in, batchSize, cublasHandle, cudnnHandle);

  //output layer - feed through
  output_layer.feedThroughLayer(hidden_layer.dev_a, output_layer.in, batchSize, cublasHandle, cudnnHandle);

  //copy hidden data back to host
  hidden_layer.copy_from_device();
  //free device pointers
  hidden_layer.free_device_ptr();

  //copy output data back to host
  output_layer.copy_from_device();
  //free device pointers
  output_layer.free_device_ptr();

  checkCudaErrors(hipFree(dev_data));

  checkCudaErrors(hipDeviceSynchronize());

  return output_layer.a;
}

double cuFFNetwork::backPropagate(float *correct_out) {

  checkCudaErrors(hipSetDevice(gpuid));
  float one = 1.0f, zero = 0.0f;

  hidden_layer.copy_to_device();
  output_layer.copy_to_device();

  float *dev_loss;
  checkCudaErrors(hipMalloc(&dev_loss, output_layer.out * sizeof(float)));
  checkCudaErrors(hipMemcpyAsync(dev_loss, output_layer.dev_a, output_layer.out * sizeof(float), hipMemcpyDeviceToDevice));

  float *dev_correct;
  checkCudaErrors(hipMalloc(&dev_correct, output_layer.out * sizeof(float)));
  checkCudaErrors(hipMemcpyAsync(dev_correct, correct_out, output_layer.out * sizeof(float), hipMemcpyHostToDevice));

  //compute error at the last layer - need to update this probably
  costFunc<<<RoundUp(batchSize, BW),BW>>>(dev_loss, output_layer.out, batchSize, dev_correct);

  //think about activationBackward function call carefully - it's gotta be simple-ish
//  checkCUDNN(hipdnnActivationBackward(cudnnHandle, output_layer.activation,
//                                     &one, output_layer.layerTensor, output_layer.dev_a, output_layer.layerTensor, dev_loss, ))

  Eigen::VectorXf readin (output_layer.out);
  checkCudaErrors(hipMemcpyAsync(readin.data(), dev_loss, output_layer.out * sizeof(float), hipMemcpyDeviceToHost));

  std::cout << readin << std::endl;

}
