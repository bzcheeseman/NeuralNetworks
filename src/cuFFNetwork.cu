//
// Created by Aman LaChapelle on 9/21/16.
//
// NeuralNetworks
// Copyright (C) 2016  Aman LaChapelle
//
// Full license at NeuralNetworks/LICENSE.txt
//

/*
    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "../include/cuFFNetwork.h"

cuFFNetwork::cuFFNetwork(int gpuid, int batchSize, cuLayer& hidden_layer, cuLayer& output_layer):
        gpuid(gpuid), hidden_layer(hidden_layer), output_layer(output_layer) {

  //set up device
  checkCudaErrors(hipSetDevice(gpuid));
  checkCudaErrors(hipblasCreate(&cublasHandle));
  checkCUDNN(hipdnnCreate(&cudnnHandle));

  checkCUDNN(hipdnnCreateTensorDescriptor(&input_data)); // init tensor for input data
  checkCUDNN(hipdnnCreateTensorDescriptor(&hiddenTensor)); // init tensor for hidden layer
  checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor)); // init tensor for output layer

  checkCUDNN(hipdnnCreateActivationDescriptor(&hidden_activations)); // init hidden layer activations
  checkCUDNN(hipdnnCreateActivationDescriptor(&output_activations)); // init output layer activations

  checkCUDNN(hipdnnSetTensor4dDescriptor(hiddenTensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, batchSize, hidden_layer.out, 1, 1));
  checkCUDNN(hipdnnSetTensor4dDescriptor(outputTensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, batchSize, output_layer.out, 1, 1));

  if (activation_func == ReLU){
    checkCUDNN(hipdnnSetActivationDescriptor(hidden_activations, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0.0));
  }
  else if (activation_func == Tanh){
    checkCUDNN(hipdnnSetActivationDescriptor(hidden_activations, HIPDNN_ACTIVATION_TANH, HIPDNN_PROPAGATE_NAN, 0.0));
  }
  else if (activation_func == Sigmoid){
    checkCUDNN(hipdnnSetActivationDescriptor(hidden_activations, HIPDNN_ACTIVATION_SIGMOID, HIPDNN_PROPAGATE_NAN, 0.0));
  }


  checkCUDNN(hipdnnSetActivationDescriptor(output_activations, HIPDNN_ACTIVATION_SIGMOID, HIPDNN_PROPAGATE_NAN, 0.0));

}

cuFFNetwork::~cuFFNetwork() {

  checkCudaErrors(hipSetDevice(gpuid));
  checkCudaErrors(hipblasDestroy(cublasHandle));
  checkCUDNN(hipdnnDestroy(cudnnHandle));

  checkCUDNN(hipdnnDestroyTensorDescriptor(input_data));
  checkCUDNN(hipdnnDestroyTensorDescriptor(hiddenTensor));
  checkCUDNN(hipdnnDestroyTensorDescriptor(outputTensor));

  checkCUDNN(hipdnnDestroyActivationDescriptor(hidden_activations));
  checkCUDNN(hipdnnDestroyActivationDescriptor(output_activations));

}

//void cuFFNetwork::feedForward(double *data, double *hidden, double* hiddenact,
//                                            double *output, double *result,
//                                            double *hiddenWeight, double *hiddenBias,
//                                            double *outputWeight, double *outputBias,
//                              double *ones) {
//  double alpha = 1.0;
//  double beta = 0.0;
//
//  checkCudaErrors(hipSetDevice(gpuid));
//
//  /*
//   * C = alpha * op(A) * op(B) + beta*C
//   *
//   * hipblasSgemm(handler, op1, op2, rows of op(A) = rows of C, cols of op(B) = cols of C, cols of op(A) = rows of op(B), scalar alpha,
//   *             input array A, leading dimension of A (rows of A if op(A) = A, cols of A if op(A) = A.transpose() or A.dagger(),
//   *             input array B, leading dimension of A (rows of B if op(B) = B, cols of B if op(B) = B.transpose() or B.dagger(),
//   *             scalar beta, input array C dims = rows of op(A) x cols of op(B), leading dimension of C (rows of op(A))
//   */
//
//  //Hidden layer forward propagation
//  //weights
//
//
//}
//

