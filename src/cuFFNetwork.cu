#include "hip/hip_runtime.h"
//
// Created by Aman LaChapelle on 9/21/16.
//
// NeuralNetworks
// Copyright (C) 2016  Aman LaChapelle
//
// Full license at NeuralNetworks/LICENSE.txt
//

/*
    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "../include/cuFFNetwork.hpp"

/*******************************************
 * cuLayer
 *******************************************/

cuLayer::cuLayer(int in, int out) : in(in), out(out) {
  w = Eigen::MatrixXf(out, in);
  b = Eigen::VectorXf(out);
  z = Eigen::VectorXf(out);
  a = Eigen::VectorXf(out);

  checkCudaErrors(hipSetDevice(0));

  float *devicedata;
  float mean = (float)0.0;
  float stddev = (float)(1.0/(float)sqrt( (float)in ));

  hiprandGenerator_t gen;
  //create generator
  checkCurandErrors(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
  //set generator seed
  auto now = std::chrono::high_resolution_clock::now();
  std::uint64_t nanos = std::chrono::duration_cast<std::chrono::nanoseconds>(now.time_since_epoch()).count();
  checkCurandErrors(hiprandSetPseudoRandomGeneratorSeed(gen, nanos)); //set seed here

  checkCudaErrors(hipMalloc(&devicedata, (in*out)*sizeof(float))); //malloc size of weights
  checkCurandErrors(hiprandGenerateNormal(gen, devicedata, (std::size_t)(in*out + (in*out)%2), mean, stddev)); //generate numbers
  checkCudaErrors(hipMemcpy(w.data(), devicedata, (in*out)*sizeof(float), hipMemcpyDeviceToHost)); //copy it back
  checkCudaErrors(hipFree(devicedata)); //free pointer to realloc

  checkCudaErrors(hipMalloc((void **)&devicedata, (out)*sizeof(float))); //realloc for biases
  checkCurandErrors(hiprandGenerateNormal(gen, devicedata, (std::size_t)(out + out%2), mean, stddev)); //generate numbers
  checkCudaErrors(hipMemcpy(b.data(), devicedata, (out)*sizeof(float), hipMemcpyDeviceToHost)); //copy it back

  float *dev_z, *dev_a;

  checkCudaErrors(hipMalloc(&dev_z, out * sizeof(float)));
  checkCudaErrors(hipMemset(dev_z, 0.0f, out*sizeof(float)));
  checkCudaErrors(hipMemcpy(z.data(), dev_z, out*sizeof(float), hipMemcpyDeviceToHost));

  checkCudaErrors(hipMalloc(&dev_a, out * sizeof(float)));
  checkCudaErrors(hipMemset(dev_a, 0.0f, out*sizeof(float)));
  checkCudaErrors(hipMemcpy(a.data(), dev_a, out*sizeof(float), hipMemcpyDeviceToHost));


  checkCudaErrors(hipFree(devicedata)); //free pointer
  checkCudaErrors(hipFree(dev_z));
  checkCudaErrors(hipFree(dev_a));
  checkCurandErrors(hiprandDestroyGenerator(gen));

  checkCudaErrors(hipDeviceSynchronize());
}

std::ostream &operator<<(std::ostream &out, cuLayer &layer) {
  out << "Inputs: " << layer.in << " Outputs: " << layer.out << std::endl;
  out << "==========Weights==========\n" << layer.w << std::endl;
  out << "\n==========Bias==========\n"<< layer.b << std::endl;
  out << "\n==========Z==========\n"<< layer.z << std::endl;
  out << "\n==========Activations==========\n"<< layer.a << std::endl;
  return out;
}



/*******************************************
 * cuFFNetwork
 *******************************************/

cuFFNetwork::cuFFNetwork(int gpuid, int batchSize, cuLayer& hidden_layer, cuLayer& output_layer):
        gpuid(gpuid), batchSize(batchSize), hidden_layer(hidden_layer), output_layer(output_layer) {

  //set up device
  checkCudaErrors(hipSetDevice(gpuid));
  checkCudaErrors(hipblasCreate(&cublasHandle));
  checkCUDNN(hipdnnCreate(&cudnnHandle));

  activation_func = Tanh;

  checkCUDNN(hipdnnCreateTensorDescriptor(&input_data)); // init tensor for input data
  checkCUDNN(hipdnnCreateTensorDescriptor(&(hidden_layer.weight))); // init weight tensor for hidden layer
  checkCUDNN(hipdnnCreateTensorDescriptor(&(hidden_layer.bias))); // init bias tensor for hidden layer
  checkCUDNN(hipdnnCreateTensorDescriptor(&(hidden_layer.zs))); // init z tensor for hiddens
  checkCUDNN(hipdnnCreateTensorDescriptor(&(hidden_layer.as))); // init a tensor for hiddens

  checkCUDNN(hipdnnCreateTensorDescriptor(&(output_layer.weight))); // init weight tensor for output layer
  checkCUDNN(hipdnnCreateTensorDescriptor(&(output_layer.bias))); // init bias tensor for output layer
  checkCUDNN(hipdnnCreateTensorDescriptor(&(output_layer.zs))); // init z tensor for hiddens
  checkCUDNN(hipdnnCreateTensorDescriptor(&(output_layer.as))); // init a tensor for hiddens

  checkCUDNN(hipdnnCreateActivationDescriptor(&(hidden_layer.activation))); // init hidden layer activations
  checkCUDNN(hipdnnCreateActivationDescriptor(&(output_layer.activation))); // init output layer activations

  checkCUDNN(hipdnnSetTensor4dDescriptor(hidden_layer.weight, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchSize, hidden_layer.out, 1, 1));
  checkCUDNN(hipdnnSetTensor4dDescriptor(hidden_layer.bias, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchSize, hidden_layer.out, 1, 1));
  checkCUDNN(hipdnnSetTensor4dDescriptor(hidden_layer.zs, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchSize, hidden_layer.out, 1, 1));
  checkCUDNN(hipdnnSetTensor4dDescriptor(hidden_layer.as, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchSize, hidden_layer.out, 1, 1));

  checkCUDNN(hipdnnSetTensor4dDescriptor(output_layer.weight, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchSize, output_layer.out, 1, 1));
  checkCUDNN(hipdnnSetTensor4dDescriptor(output_layer.bias, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchSize, output_layer.out, 1, 1));
  checkCUDNN(hipdnnSetTensor4dDescriptor(output_layer.zs, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchSize, output_layer.out, 1, 1));
  checkCUDNN(hipdnnSetTensor4dDescriptor(output_layer.as, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchSize, output_layer.out, 1, 1));

  if (activation_func == ReLU){
    checkCUDNN(hipdnnSetActivationDescriptor(hidden_layer.activation, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0.0));
  }
  else if (activation_func == Tanh){
    checkCUDNN(hipdnnSetActivationDescriptor(hidden_layer.activation, HIPDNN_ACTIVATION_TANH, HIPDNN_PROPAGATE_NAN, 0.0));
  }
  else if (activation_func == Sigmoid){
    checkCUDNN(hipdnnSetActivationDescriptor(hidden_layer.activation, HIPDNN_ACTIVATION_SIGMOID, HIPDNN_PROPAGATE_NAN, 0.0));
  }


  checkCUDNN(hipdnnSetActivationDescriptor(output_layer.activation, HIPDNN_ACTIVATION_SIGMOID, HIPDNN_PROPAGATE_NAN, 0.0));

  checkCUDNN(hipdnnCreateOpTensorDescriptor(&mult));
  checkCUDNN(hipdnnCreateOpTensorDescriptor(&add));

  checkCUDNN(hipdnnSetOpTensorDescriptor(mult, HIPDNN_OP_TENSOR_MUL, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN));
  checkCUDNN(hipdnnSetOpTensorDescriptor(add, HIPDNN_OP_TENSOR_ADD, HIPDNN_DATA_FLOAT, HIPDNN_PROPAGATE_NAN));

}

cuFFNetwork::~cuFFNetwork() {

  checkCudaErrors(hipSetDevice(gpuid));
  checkCudaErrors(hipblasDestroy(cublasHandle));
  checkCUDNN(hipdnnDestroy(cudnnHandle));

  checkCUDNN(hipdnnDestroyTensorDescriptor(input_data));

}

Eigen::VectorXf cuFFNetwork::feedForward(float *data) {
  checkCudaErrors(hipSetDevice(gpuid));

  float one = 1.0f, zero = 0.0f;

  //need to copy all these vectors into the device - call hipMemcpy, etc. for the shit to actually happen...
  //use hipMemcpyHostToDevice to copy everything in

  float *dev_hidden_w, *dev_hidden_b, *dev_hidden_z, *dev_hidden_a;

  //malloc and copy over hidden layer parameters
  checkCudaErrors(hipMalloc(&dev_hidden_w, hidden_layer.in*hidden_layer.out * sizeof(float)));
  checkCudaErrors(hipMemcpy(dev_hidden_w, &hidden_layer.w.data()[0],
                                  hidden_layer.in*hidden_layer.out * sizeof(float), hipMemcpyHostToDevice));

  checkCudaErrors(hipMalloc(&dev_hidden_b, hidden_layer.out * sizeof(float)));
  checkCudaErrors(hipMemcpy(dev_hidden_b, &hidden_layer.b.data()[0],
                                  hidden_layer.out * sizeof(float), hipMemcpyHostToDevice));

  checkCudaErrors(hipMalloc(&dev_hidden_z, hidden_layer.out * sizeof(float)));
  checkCudaErrors(hipMemcpy(dev_hidden_z, &hidden_layer.z.data()[0],
                                  hidden_layer.out * sizeof(float), hipMemcpyHostToDevice));

  checkCudaErrors(hipMalloc(&dev_hidden_a, hidden_layer.out * sizeof(float)));
  checkCudaErrors(hipMemcpy(dev_hidden_a, &hidden_layer.a.data()[0],
                                  hidden_layer.out * sizeof(float), hipMemcpyHostToDevice));

  float *dev_output_w, *dev_output_b, *dev_output_z, *dev_output_a;

  //malloc and copy over output layer parameters
  checkCudaErrors(hipMalloc(&dev_output_w, output_layer.in*output_layer.out * sizeof(float)));
  checkCudaErrors(hipMemcpy(dev_output_w, &output_layer.w.data()[0],
                                  output_layer.in*output_layer.out * sizeof(float), hipMemcpyHostToDevice));

  checkCudaErrors(hipMalloc(&dev_output_b, output_layer.out * sizeof(float)));
  checkCudaErrors(hipMemcpy(dev_output_b, &output_layer.b.data()[0],
                                  output_layer.out * sizeof(float), hipMemcpyHostToDevice));

  checkCudaErrors(hipMalloc(&dev_output_z, output_layer.out * sizeof(float)));
  checkCudaErrors(hipMemcpy(dev_output_z, &output_layer.z.data()[0],
                             output_layer.out * sizeof(float), hipMemcpyHostToDevice));

  checkCudaErrors(hipMalloc(&dev_output_a, output_layer.out * sizeof(float)));
  checkCudaErrors(hipMemcpy(dev_output_a, &output_layer.a.data()[0],
                             output_layer.out * sizeof(float), hipMemcpyHostToDevice));


  float *dev_data;
  checkCudaErrors(hipMalloc(&dev_data, hidden_layer.in * sizeof(float)));
  checkCudaErrors(hipMemcpy(dev_data, &data[0], batchSize*hidden_layer.in * sizeof(float), hipMemcpyHostToDevice));


  float *ones;
  checkCudaErrors(hipMalloc(&ones, batchSize * sizeof(float)));
  checkCudaErrors(hipMemset(ones, 1.0f, batchSize * sizeof(float)));

  //! CHECK COMPUTATIONS - NOT TOTALLY CONVINCED THEY'RE RIGHT - esp. cublas

  //hidden layer

  checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                              hidden_layer.out, batchSize, hidden_layer.out,
                              &one, dev_hidden_w, hidden_layer.out, dev_data, hidden_layer.in,
                              &zero, dev_hidden_z, hidden_layer.out));

  checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                              hidden_layer.out, batchSize, 1,
                              &one, dev_hidden_b, hidden_layer.out, ones, 1,
                              &one, dev_hidden_z, hidden_layer.out));

  checkCUDNN(hipdnnActivationForward(cudnnHandle, hidden_layer.activation,
                                    &one, hidden_layer.zs, dev_hidden_z,
                                    &zero, hidden_layer.as, dev_hidden_a));

  //output layer

  checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                              output_layer.out, batchSize, output_layer.in,
                              &one, dev_output_w, output_layer.out, dev_hidden_a, output_layer.in,
                              &zero, dev_output_z, output_layer.out));

  checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                              output_layer.out, batchSize, 1,
                              &one, dev_output_b, output_layer.out, ones, 1,
                              &one, dev_output_z, output_layer.out));

  checkCUDNN(hipdnnActivationForward(cudnnHandle, output_layer.activation,
                                    &one, output_layer.zs, dev_output_z,
                                    &zero, output_layer.as, dev_output_a));

  //copy hidden data back to host
  checkCudaErrors(hipMemcpy(hidden_layer.z.data(), dev_hidden_z, hidden_layer.out*sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(hidden_layer.a.data(), dev_hidden_a, hidden_layer.out*sizeof(float), hipMemcpyDeviceToHost));

  //copy output data back to host
  checkCudaErrors(hipMemcpy(output_layer.z.data(), dev_output_z, output_layer.out*sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(output_layer.a.data(), dev_output_a, output_layer.out*sizeof(float), hipMemcpyDeviceToHost));

  checkCudaErrors(hipFree(ones));
  checkCudaErrors(hipFree(dev_data));
  checkCudaErrors(hipFree(dev_hidden_w));
  checkCudaErrors(hipFree(dev_hidden_b));
  checkCudaErrors(hipFree(dev_hidden_z));
  checkCudaErrors(hipFree(dev_hidden_a));
  checkCudaErrors(hipFree(dev_output_w));
  checkCudaErrors(hipFree(dev_output_b));
  checkCudaErrors(hipFree(dev_output_z));
  checkCudaErrors(hipFree(dev_output_a));

  checkCudaErrors(hipDeviceSynchronize());

  return output_layer.a;
}