#include "hip/hip_runtime.h"
//
// Created by Aman LaChapelle on 9/21/16.
//
// NeuralNetworks
// Copyright (C) 2016  Aman LaChapelle
//
// Full license at NeuralNetworks/LICENSE.txt
//

/*
    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "../include/cuFFNetwork.hpp"

static inline unsigned int RoundUp(unsigned int numerator, unsigned int denominator)
{
  return (numerator + denominator - 1) / denominator;
}

static inline void printDeviceVector(float *dev_vector, int size){
  Eigen::VectorXf vec (size);

  checkCudaErrors(hipMemcpy(vec.data(), dev_vector, size * sizeof(float), hipMemcpyDeviceToHost));

  std::cout << vec << std::endl;

}


/*******************************************
 * cuLayer
 *******************************************/

cuFFLayer::cuFFLayer(int in, int out, int gpuid, int batchSize) : in(in), out(out), gpuid(gpuid), batchSize(batchSize) {
  w = Eigen::MatrixXf(out, in);
  b = Eigen::VectorXf(out);

  z = Eigen::MatrixXf(out, batchSize);
  a = Eigen::MatrixXf(out, batchSize);
  delta = Eigen::MatrixXf(out, batchSize);

  dw = Eigen::MatrixXf(out, in);
  db = Eigen::VectorXf(out);

  checkCudaErrors(hipSetDevice(gpuid));

  float *devicedata;
  float mean = (float)0.0;
  float stddev = (float)(1.0/(float)sqrt( (float)in ));

  hiprandGenerator_t gen;
  //create generator
  checkCurandErrors(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
  //set generator seed
  auto now = std::chrono::high_resolution_clock::now();
  std::uint64_t nanos = std::chrono::duration_cast<std::chrono::nanoseconds>(now.time_since_epoch()).count();
  checkCurandErrors(hiprandSetPseudoRandomGeneratorSeed(gen, nanos)); //set seed here

  checkCudaErrors(hipMalloc(&devicedata, (in*out)*sizeof(float))); //malloc size of weights
  checkCurandErrors(hiprandGenerateNormal(gen, devicedata, (std::size_t)(in*out + (in*out)%2), mean, stddev)); //generate numbers
  checkCudaErrors(hipMemcpy(w.data(), devicedata, (in*out)*sizeof(float), hipMemcpyDeviceToHost)); //copy it back
  checkCudaErrors(hipFree(devicedata)); //free pointer to realloc

  checkCudaErrors(hipMalloc((void **)&devicedata, (out)*sizeof(float))); //realloc for biases
  checkCurandErrors(hiprandGenerateNormal(gen, devicedata, (std::size_t)(out + out%2), mean, stddev)); //generate numbers
  checkCudaErrors(hipMemcpy(b.data(), devicedata, (out)*sizeof(float), hipMemcpyDeviceToHost)); //copy it back

  float *dev_z, *dev_a;

  checkCudaErrors(hipMalloc(&dev_z, out*batchSize * sizeof(float)));
  checkCudaErrors(hipMemset(dev_z, 0.0f, out*batchSize*sizeof(float)));
  checkCudaErrors(hipMemcpy(z.data(), dev_z, out*batchSize*sizeof(float), hipMemcpyDeviceToHost));

  checkCudaErrors(hipMalloc(&dev_a, out*batchSize * sizeof(float)));
  checkCudaErrors(hipMemset(dev_a, 0.0f, out*batchSize*sizeof(float)));
  checkCudaErrors(hipMemcpy(a.data(), dev_a, out*batchSize*sizeof(float), hipMemcpyDeviceToHost));


  checkCudaErrors(hipFree(devicedata)); //free pointer
  checkCudaErrors(hipFree(dev_z));
  checkCudaErrors(hipFree(dev_a));
  checkCurandErrors(hiprandDestroyGenerator(gen));

  checkCudaErrors(hipDeviceSynchronize());
}

cuFFLayer::~cuFFLayer() {
  checkCudaErrors(hipSetDevice(gpuid));
  checkCUDNN(hipdnnDestroyTensorDescriptor(layerTensor));
  checkCUDNN(hipdnnDestroyActivationDescriptor(activation));
}

void cuFFLayer::initTensor() {

  checkCudaErrors(hipSetDevice(gpuid));

  checkCUDNN(hipdnnCreateTensorDescriptor(&(layerTensor))); // init tensor for this layer

  checkCUDNN(hipdnnSetTensor4dDescriptor(layerTensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchSize, out, 1, 1));

}

void cuFFLayer::setActivation(hipdnnActivationMode_t cudnnActivationFunc) {
  checkCudaErrors(hipSetDevice(gpuid));

  checkCUDNN(hipdnnCreateActivationDescriptor(&(activation)));
  checkCUDNN(hipdnnSetActivationDescriptor(activation, cudnnActivationFunc, HIPDNN_PROPAGATE_NAN, 0.0));
}

void cuFFLayer::copy_to_device() {

  checkCudaErrors(hipSetDevice(gpuid));

  checkCudaErrors(hipMalloc(&dev_w, in*out * sizeof(float)));
  checkCudaErrors(hipMemcpyAsync(dev_w, &w.data()[0], in*out * sizeof(float), hipMemcpyHostToDevice));

  checkCudaErrors(hipMalloc(&dev_b, out * sizeof(float)));
  checkCudaErrors(hipMemcpyAsync(dev_b, &b.data()[0], out * sizeof(float), hipMemcpyHostToDevice));

  checkCudaErrors(hipMalloc(&dev_z, out*batchSize * sizeof(float)));
  checkCudaErrors(hipMemcpyAsync(dev_z, &z.data()[0], out*batchSize * sizeof(float), hipMemcpyHostToDevice));

  checkCudaErrors(hipMalloc(&dev_a, out*batchSize * sizeof(float)));
  checkCudaErrors(hipMemcpyAsync(dev_a, &a.data()[0], out*batchSize * sizeof(float), hipMemcpyHostToDevice));

}

void cuFFLayer::copy_from_device() {

  checkCudaErrors(hipSetDevice(gpuid));

  checkCudaErrors(hipMemcpyAsync(w.data(), dev_w, out*in*sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpyAsync(b.data(), dev_b, out*sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpyAsync(z.data(), dev_z, out*batchSize*sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpyAsync(a.data(), dev_a, out*batchSize*sizeof(float), hipMemcpyDeviceToHost));

}

void cuFFLayer::free_device_ptr() {

  checkCudaErrors(hipSetDevice(gpuid));

  checkCudaErrors(hipFree(dev_w));
  checkCudaErrors(hipFree(dev_b));
  checkCudaErrors(hipFree(dev_z));
  checkCudaErrors(hipFree(dev_a));

//  checkCudaErrors(hipFree(dev_gradient));
//  checkCudaErrors(hipFree(dCdw));
//  checkCudaErrors(hipFree(dCdb));

}

void cuFFLayer::feedThroughLayer(float *device_ptr_input, hipblasHandle_t cublasHandle, hipdnnHandle_t cudnnHandle) {

  checkCudaErrors(hipSetDevice(gpuid));

  float one = 1.0f, zero = 0.0f;

  thrust::device_vector<float> ones(batchSize, 1.0f);

  checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                              out, in, batchSize,
                              &one, dev_w, out, device_ptr_input, in,
                              &zero, dev_z, out));

  checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                              out, batchSize, 1,
                              &one, dev_b, out, thrust::raw_pointer_cast(&ones[0]), 1,
                              &one, dev_z, out));

  checkCUDNN(hipdnnActivationForward(cudnnHandle, activation,
                                    &one, layerTensor, dev_z,
                                    &zero, layerTensor, dev_a)); //apply activation within the layer, before giving away output
}

void cuFFLayer::init_gradient() {

  checkCudaErrors(hipSetDevice(gpuid));

  //init gradient tensor
  checkCUDNN(hipdnnCreateTensorDescriptor(&(deltaTensor))); // init tensor for this layer
  checkCUDNN(hipdnnSetTensor4dDescriptor(deltaTensor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batchSize, out, 1, 1));

  //now copy over the array to the device
  checkCudaErrors(hipMalloc(&dev_delta, out*batchSize*sizeof(float)));
  checkCudaErrors(hipMemset(dev_delta, 0.0f, out*batchSize*sizeof(float))); //this doesn't actually work - need to actually set
                                                                             //numbers to zero!

  checkCudaErrors(hipMalloc(&dCdw, in*out*sizeof(float)));
  checkCudaErrors(hipMemset(dCdw, 0.0f, in*out*sizeof(float)));

  checkCudaErrors(hipMalloc(&dCdb, out*sizeof(float)));
  checkCudaErrors(hipMemset(dCdb, 0.0f, out*sizeof(float)));

}

void cuFFLayer::copy_back_gradient() {

  checkCudaErrors(hipSetDevice(gpuid));

  checkCudaErrors(hipMemcpyAsync(delta.data(), dev_delta, out*batchSize*sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpyAsync(dw.data(), dCdw, in*out*sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpyAsync(db.data(), dCdb, out*sizeof(float), hipMemcpyDeviceToHost));

}

std::ostream &operator<<(std::ostream &out, cuFFLayer &layer) {
  out << "Inputs: " << layer.in << " Outputs: " << layer.out << std::endl;
  out << "==========Weights==========\n" << layer.w << std::endl;
  out << "\n==========Bias==========\n"<< layer.b << std::endl;
  out << "\n==========Z==========\n"<< layer.z << std::endl;
  out << "\n==========Activations==========\n"<< layer.a << std::endl;
  return out;
}


/*******************************************
 * cuFFNetwork
 *******************************************/

cuFFNetwork::cuFFNetwork(int gpuid, int batchSize, cuFFLayer& hidden_layer, cuFFLayer& output_layer):
        gpuid(gpuid), batchSize(batchSize), hidden_layer(hidden_layer), output_layer(output_layer) {

  //set up device
  checkCudaErrors(hipSetDevice(gpuid));

  checkCudaErrors(hipblasCreate(&cublasHandle));
  checkCUDNN(hipdnnCreate(&cudnnHandle));

  activation_func = Sigmoid;

  checkCUDNN(hipdnnCreateTensorDescriptor(&input_data)); // init tensor for input data

  this->hidden_layer.initTensor();

  this->output_layer.initTensor();

  if (activation_func == ReLU){
    this->hidden_layer.setActivation(HIPDNN_ACTIVATION_RELU);
  }
  else if (activation_func == Tanh){
    this->hidden_layer.setActivation(HIPDNN_ACTIVATION_TANH);
  }
  else if (activation_func == Sigmoid){
    this->hidden_layer.setActivation(HIPDNN_ACTIVATION_SIGMOID);
  }


  this->output_layer.setActivation(HIPDNN_ACTIVATION_SIGMOID);

}

cuFFNetwork::~cuFFNetwork() {

  checkCudaErrors(hipSetDevice(gpuid));

  checkCudaErrors(hipblasDestroy(cublasHandle));
  checkCUDNN(hipdnnDestroy(cudnnHandle));

  checkCUDNN(hipdnnDestroyTensorDescriptor(input_data));

}

Eigen::VectorXf cuFFNetwork::feedForward(float *data) {
  checkCudaErrors(hipSetDevice(gpuid));

  hidden_layer.copy_to_device();

  output_layer.copy_to_device();

  //Copy data to device to pass to the hidden layer
  float *dev_data;
  checkCudaErrors(hipMalloc(&dev_data, batchSize * hidden_layer.in * sizeof(float)));
  checkCudaErrors(hipMemcpy(dev_data, &data[0], batchSize * hidden_layer.in * sizeof(float), hipMemcpyHostToDevice));

  //hidden layer - feed through
  hidden_layer.feedThroughLayer(dev_data, cublasHandle, cudnnHandle);

  //output layer - feed through
  output_layer.feedThroughLayer(hidden_layer.dev_a, cublasHandle, cudnnHandle);

  //copy hidden data back to host
  hidden_layer.copy_from_device();
  //free device pointers
  hidden_layer.free_device_ptr();

  //copy output data back to host
  output_layer.copy_from_device();
  //free device pointers
  output_layer.free_device_ptr();

  checkCudaErrors(hipFree(dev_data));

  checkCudaErrors(hipDeviceSynchronize());

  return output_layer.a;
}

double cuFFNetwork::backPropagate(float *inputs, float *correct_out, int iterations) {

  checkCudaErrors(hipSetDevice(gpuid));
  float one = 1.0f, zero = 0.0f;
  float eta = 0.05f;

  thrust::device_vector<float> ones(batchSize, 1.0f);

  hidden_layer.copy_to_device();
  output_layer.copy_to_device();

  output_layer.init_gradient();
  hidden_layer.init_gradient();

  float *dev_inputs;
  checkCudaErrors(hipMalloc(&dev_inputs, batchSize * hidden_layer.in * sizeof(float)));
  checkCudaErrors(hipMemcpyAsync(dev_inputs, &(inputs[0]), batchSize * hidden_layer.in * sizeof(float), hipMemcpyHostToDevice));

  thrust::device_vector<float> input_activations(batchSize*hidden_layer.in, 0.0f);

  checkCUDNN(hipdnnActivationForward(cudnnHandle, hidden_layer.activation,
                                    &one, hidden_layer.layerTensor, dev_inputs,
                                    &zero, hidden_layer.layerTensor, thrust::raw_pointer_cast(&input_activations[0])));

  float *dev_correct;
  checkCudaErrors(hipMalloc(&dev_correct, batchSize * output_layer.out * sizeof(float)));
  checkCudaErrors(hipMemcpyAsync(dev_correct, &(correct_out[0]), batchSize * output_layer.out * sizeof(float), hipMemcpyHostToDevice));

  float *dev_cost;
  checkCudaErrors(hipMalloc(&dev_cost, batchSize * output_layer.out * sizeof(float)));

  for (int i = 0; i < iterations; i++){

    //feed forward
    hidden_layer.feedThroughLayer(dev_inputs, cublasHandle, cudnnHandle);
    output_layer.feedThroughLayer(hidden_layer.dev_a, cublasHandle, cudnnHandle);

    checkCudaErrors(hipblasScopy(cublasHandle, batchSize * output_layer.out, output_layer.dev_a, 1, dev_cost, 1));

    costFunc<<<RoundUp(batchSize, BW), BW>>>(dev_cost, output_layer.out, batchSize, dev_correct); //costfunc

    checkCUDNN(hipdnnActivationBackward(cudnnHandle, output_layer.activation,
                                       &one, output_layer.layerTensor, output_layer.dev_a,
                                       output_layer.layerTensor, dev_cost,
                                       output_layer.layerTensor, output_layer.dev_z,
                                       &zero, output_layer.deltaTensor, output_layer.dev_delta));

    //compute bias gradient (collapse along one axis)
    checkCudaErrors(hipblasSgemv(cublasHandle, HIPBLAS_OP_N, output_layer.out, batchSize,
                                &one, output_layer.dev_delta, output_layer.out, thrust::raw_pointer_cast(&ones[0]), 1,
                                &zero, output_layer.dCdb, 1));

    //compute weights gradient
    checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, output_layer.out, hidden_layer.out, batchSize,
                                &one, output_layer.dev_delta, output_layer.out, hidden_layer.dev_a, hidden_layer.out,
                                &zero, output_layer.dCdw, output_layer.out));

    checkCudaErrors(hipFree(dev_cost));
    checkCudaErrors(hipMalloc(&dev_cost, hidden_layer.out * batchSize * sizeof(float)));

    //compute loss for hidden layer - gotta check this guy
    checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, hidden_layer.out, hidden_layer.in, batchSize,
                                &one, output_layer.dev_w, hidden_layer.out, output_layer.dev_delta, hidden_layer.in,
                                &zero, dev_cost, hidden_layer.out));

    //backward through hidden layer
    checkCUDNN(hipdnnActivationBackward(cudnnHandle, hidden_layer.activation,
                                       &one, hidden_layer.layerTensor, hidden_layer.dev_a,
                                       hidden_layer.layerTensor, dev_cost,
                                       hidden_layer.layerTensor, hidden_layer.dev_z,
                                       &zero, hidden_layer.deltaTensor, hidden_layer.dev_delta));


    //bias and weights gradient
    checkCudaErrors(hipblasSgemv(cublasHandle, HIPBLAS_OP_N, hidden_layer.out, batchSize,
                                &one, hidden_layer.dev_delta, hidden_layer.out, thrust::raw_pointer_cast(&ones[0]), 1,
                                &zero, hidden_layer.dCdb, 1));

    checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, hidden_layer.out, hidden_layer.in, batchSize,
                                &one, hidden_layer.dev_delta, hidden_layer.out, dev_inputs, hidden_layer.in,
                                &zero, hidden_layer.dCdw, hidden_layer.out));

    //update output layer
    checkCudaErrors(hipblasSaxpy(cublasHandle, output_layer.out,
                                &eta, output_layer.dCdb, 1, output_layer.dev_b, 1));

    checkCudaErrors(hipblasSgeam(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, output_layer.out, output_layer.in,
                                &eta, output_layer.dCdw, output_layer.out,
                                &one, output_layer.dev_w, output_layer.out,
                                output_layer.dev_w, output_layer.out));

    //update hidden layer
    checkCudaErrors(hipblasSaxpy(cublasHandle, hidden_layer.out,
                                &eta, hidden_layer.dCdb, 1, hidden_layer.dev_b, 1));

    checkCudaErrors(hipblasSgeam(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, hidden_layer.out, hidden_layer.in,
                                &eta, hidden_layer.dCdw, hidden_layer.out,
                                &one, hidden_layer.dev_w, hidden_layer.out,
                                hidden_layer.dev_w, hidden_layer.out));
  }

  hidden_layer.copy_back_gradient();
  output_layer.copy_back_gradient();

  hidden_layer.copy_from_device();
  output_layer.copy_from_device();


  return 1.0;
}
