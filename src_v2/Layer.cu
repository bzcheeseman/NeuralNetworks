#include "hip/hip_runtime.h"
//
// Created by Aman LaChapelle on 10/16/16.
//
// NeuralNetworks
// Copyright (c) 2016 Aman LaChapelle
// Full license at NeuralNetworks/LICENSE.txt
//

/*
    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */


#include "../include_v2/Layer.hpp"

FFLayer::FFLayer(int in, int out, unsigned gpuid, int batchSize) :
        inputs(in), outputs(out), gpuid(gpuid), batchSize(batchSize) {

  checkCudaErrors(hipblasCreate(&cublasHandle));
  checkCUDNN(hipdnnCreate(&cudnnHandle));

  cpu_w = Eigen::MatrixXf(outputs, inputs);
  device_w = thrust::device_vector<float>(outputs*inputs);
  raw_device_w = thrust::raw_pointer_cast(&(device_w[0]));

  cpu_b = Eigen::VectorXf(out);
  device_b = thrust::device_vector<float>(outputs);
  raw_device_b = thrust::raw_pointer_cast(&(device_b[0]));

  ones = thrust::device_vector<float>(batchSize, 1.0f);
  raw_ones = thrust::raw_pointer_cast(&ones[0]);

  cpu_z = Eigen::MatrixXf(outputs, batchSize);
  device_z = thrust::device_vector<float>(outputs*batchSize);
  raw_device_z = thrust::raw_pointer_cast(&(device_z[0]));

  cpu_a = Eigen::MatrixXf(outputs, batchSize);
  device_a = thrust::device_vector<float>(outputs*batchSize);
  raw_device_a = thrust::raw_pointer_cast(&(device_a[0]));

  checkCudaErrors(hipSetDevice(gpuid));

  checkCUDNN(hipdnnCreateActivationDescriptor(&(layerActivation)));
  checkCUDNN(hipdnnSetActivationDescriptor(layerActivation, HIPDNN_ACTIVATION_SIGMOID, HIPDNN_PROPAGATE_NAN, 0.0));

  float mean = (float)0.0;
  float stddev = (float)(1.0 /sqrt( (float)in ));

  hiprandGenerator_t gen; //create generator
  checkCurandErrors(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT)); //set generator seed

  auto now = std::chrono::high_resolution_clock::now();
  std::uint64_t nanos = std::chrono::duration_cast<std::chrono::nanoseconds>(now.time_since_epoch()).count();
  checkCurandErrors(hiprandSetPseudoRandomGeneratorSeed(gen, nanos)); //set seed from chrono::now()

  checkCurandErrors(hiprandGenerateNormal(gen, raw_device_w, (std::size_t)(outputs*inputs + (outputs*inputs)%2), mean, stddev)); //generate numbers
  checkCudaErrors(hipMemcpyAsync(cpu_w.data(), raw_device_w, (outputs*inputs)*sizeof(float), hipMemcpyDeviceToHost)); //copy it back

  checkCurandErrors(hiprandGenerateNormal(gen, raw_device_b, (std::size_t)(outputs + outputs%2), mean, stddev)); //generate numbers
  checkCudaErrors(hipMemcpyAsync(cpu_b.data(), raw_device_b, (outputs)*sizeof(float), hipMemcpyDeviceToHost)); //copy it back
}

Tensor FFLayer::feedThroughLayer(Tensor &in) {

  checkCudaErrors(hipSetDevice(gpuid));

  //variables for mixing
  float one = 1.0f, zero = 0.0f;

  //init output tensor
  Tensor out (batchSize, outputs, 1, 1, gpuid);

  //Multiply by our w vector
  checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                              outputs, inputs, batchSize,
                              &one, raw_device_w, outputs, in.raw_device_data, inputs,
                              &zero, raw_device_z, outputs));

  //Add bias
  checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                              outputs, batchSize, 1,
                              &one, raw_device_b, outputs, raw_ones, 1,
                              &one, raw_device_z, outputs));

  //Activate - we're transforming the tensor to the Cout size in previous steps so we use that tensor descriptor
  checkCUDNN(hipdnnActivationForward(cudnnHandle, layerActivation,
                                    &one, out.TensorDesc, raw_device_z,
                                    &zero, out.TensorDesc, raw_device_a)); //apply activation within the layer

  out.setDeviceData(raw_device_a);

  return out;
}

void FFLayer::initBackprop() {
  cpu_delta = Eigen::MatrixXf(outputs, batchSize);
  device_delta = thrust::device_vector<float>(outputs*batchSize);
  raw_device_delta = thrust::raw_pointer_cast(&(device_delta[0]));

  cpu_dw = Eigen::MatrixXf(outputs, inputs);
  device_dw = thrust::device_vector<float>(outputs*inputs);
  raw_device_dw = thrust::raw_pointer_cast(&(device_w[0]));

  cpu_db = Eigen::VectorXf(outputs);
  device_db = thrust::device_vector<float>(outputs);
  raw_device_db = thrust::raw_pointer_cast(&(device_w[0]));
}

void FFLayer::backThroughLayer(Tensor &backward) {

  initBackprop();

  checkCudaErrors(hipSetDevice(gpuid));

  //variables for mixing
  float one = 1.0f, zero = 0.0f;

  thrust::device_vector<float> device_cost (batchSize*outputs, 0.0f);
  float *raw_device_cost = thrust::raw_pointer_cast(&device_cost[0]);

  //Cin == the output of next layer (going backwards)

  //Get in the delta from previous layer and its weights
  checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, outputs, inputs, batchSize,
                              &one, next->raw_device_w, outputs, backward.raw_device_data, inputs,
                              &zero, raw_device_cost, outputs));

  //Now we activate backward, store it all in delta
  checkCUDNN(hipdnnActivationBackward(cudnnHandle, layerActivation,
                                     &one, backward.TensorDesc, this->raw_device_a,
                                     backward.TensorDesc, raw_device_cost,
                                     backward.TensorDesc, this->raw_device_z,
                                     &zero, backward.TensorDesc, this->raw_device_delta));

  //compute bias gradient (collapse along one axis)
  checkCudaErrors(hipblasSgemv(cublasHandle, HIPBLAS_OP_N, outputs, batchSize,
                              &one, this->raw_device_delta, outputs, raw_ones, 1,
                              &zero, this->raw_device_db, 1));

  //compute weights gradient
  checkCudaErrors(hipblasSgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_T, outputs, inputs, batchSize,
                              &one, this->raw_device_delta, outputs, prev->raw_device_a, inputs,
                              &zero, this->raw_device_dw, outputs));

}
